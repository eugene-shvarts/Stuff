// Eugene Shvarts
// STA 250 | Fall 2013

//Accept-reject sampler for the truncated normal distribution (either end finite or infinite).
//Attempts naive rejection sampling until a specified number of failures, then switches to
//the method detailed in Robert (2009) (see the Lecture notes or the assignment).
//In the unlikely event that the Robert method fails, will return NA after a (different)
//specified number of failures. 

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>


extern "C"
{
__global__ void 
rtruncnorm_kernel(float *vals, int n,             //vals is the input and output
                  float *mu, float *sigma,        //these are the distribution parameters
                  float *lo, float *hi,           //these are the truncation parameters
                  int rng_a, int rng_b, int rng_c,//these are the RNG seeds
                  int maxnaive,                   //after maxnaive attempts, switch methods
                  int maxtries)                   //after maxtries attempts, cancel with error
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

    if (idx < n) {
      // Setup the RNG:
      hiprandState rng;
      hiprand_init(rng_a+idx*rng_b, rng_c, 0, &rng);

      // Sample:
      // First try naive rejection method, until maxnaive failures. 
      for (int i = 1; i < maxnaive; i = i+1) {
        vals[idx] = mu[idx] + sigma[idx] * hiprand_normal(&rng);
        if (vals[idx] > lo[idx] && vals[idx] <= hi[idx]) {
          return;
        }
      }
      // If we made it this far without a return statement, we've had maxnaive failures.
      // So, it's time to try the more sophisticated truncated sampling method cf. Robert.
      //  First we decide between two-sided and one-sided truncation. Note that there must be
      //  some truncation; otherwise it is impossible to reach this point.
      int which_side = 0;
      float offset = 0;
      
      if (isfinite(lo[idx])) {
        offset = (lo[idx]-mu[idx])/sigma[idx];
        which_side = which_side + 1;
      }
      
      if (isfinite(hi[idx])) {
        offset = (hi[idx]-mu[idx])/sigma[idx];
        which_side = which_side - 1;
      }
      //When which_side = 1, truncation is unbounded to the right. 
      //When which_side = -1, unbounded to the left.
      //When which_side = 0, bounded on both sides.
      float a_opt = 0;  //represents the optimal alpha for the exponential distribution
      float temp = 0;   //from sampling the exponential distribution by inverse CDF
      float u = 0;      //generated to compare against the inverse CDF
      
      if (which_side != 0) {  //the single-truncated case
        a_opt = 0.5 * (which_side * offset + sqrtf(powf(offset,2) + 4));
        for (int i = 1; i<maxtries; i = i+1) {
          temp = hiprand_uniform(&rng);
          vals[idx] = which_side * offset - logf(temp) / a_opt; //the exponential sample
          temp = expf(-0.5 * powf(vals[idx]-a_opt, 2)); //using overwriting to conserve space
          u = hiprand_uniform(&rng); 
          if (u <= temp) {
            //we used a standardized normal this entire time
            vals[idx] = mu[idx] + which_side*sigma[idx]*vals[idx];
            return;
          }
        } // end maxtries loop
      } // end single-truncated case
      else {  //the both-truncated-sides case
        float mu_m = (lo[idx]-mu[idx])/sigma[idx];
        float mu_p = (hi[idx]-mu[idx])/sigma[idx];
        if (mu_p < 0) {
          a_opt = powf(mu_p,2);
        }
        else if (mu_m > 0) {
          a_opt = powf(mu_m,2);
        }
        for (int i = 1; i<maxtries; i = i+1) {
          vals[idx] = (mu_p-mu_m) * hiprand_uniform(&rng) + mu_m; //uniform on the appropriate interval
          temp = expf(0.5 * (a_opt - powf(vals[idx],2)));  //choose the right branch of the function
          u = hiprand_uniform(&rng);
          if (u <= temp) {
            //we used a standardized normal this entire time
            vals[idx] = mu[idx] + sigma[idx]*vals[idx];
            return;
          }
        } // end maxtries loop
      } // end both-truncated case
      // If the code reaches this point, maxtries has been exhausted with no luck. 
    } // end if idx < n
    vals[idx] = HIP_NAN_F; //so, return NA
    return;
} // end rtruncnorm_kernel

} // END extern "C"

